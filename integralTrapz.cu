#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include "typeDefinition.cuh"
//#include "globalVariables.cuh"
#include "hostPrototypes.h"
#include "devicePrototypes.cuh"

#include "./common/CudaSafeCall.h"

extern __constant__ int nx_d, ny_d;
extern __constant__ REAL hx_d, hy_d;
extern __constant__ int tipOffsetX_d, tipOffsetY_d;
extern __constant__ float tipx0_d, tipy0_d;

__global__ void trapz_kernel(REAL *f, REAL *g, REAL *h, REAL *w,
  REAL *dot, REAL *coeffTrapz, int *tip_count, vec5dyn *tip_vector,
  int count) {

  unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;
  unsigned int stride = blockDim.x*gridDim.x;

  __shared__ REAL cache[BLOCKSIZE_1D];

  REAL temp = 0.0;

  while(index < nx_d*ny_d) {

  //   // coeffTrapz defines a circular area of coefficients
  //   temp += (f[index]*g[index] + 
  //            h[index]*w[index] )*coeffTrapz[index];

  int i = (int)index%nx_d;
  int j = (int)floorf((index/nx_d)%nx_d);
  int cx, cy;
  REAL fxg;

  int ic = i-nx_d/2;
  int jc = j-ny_d/2;

  if ( count == 0 ) {
    cx = __float2int_rn(tipx0_d-nx_d/2);
    cy = __float2int_rn(tipy0_d-ny_d/2);
  } else {
    cx = __float2int_rn(tip_vector[*tip_count-1].x-nx_d/2);
    cy = __float2int_rn(tip_vector[*tip_count-1].y-ny_d/2);
  }

  bool sc = ( (ic-cx)*(ic-cx) + (jc-cy)*(jc-cy) ) 
    < tipOffsetX_d*tipOffsetY_d ? true : false;
  bool scb = ( (ic-cx)*(ic-cx) + (jc-cy)*(jc-cy) ) 
    == tipOffsetX_d*tipOffsetY_d ? true : false;

  fxg = sc ? (f[index]*g[index] + h[index]*w[index]) : 0.0 ; 

  temp += sc ? 4.0*fxg : ( scb ? fxg : 2.0*fxg );

    index += stride;


  }

  cache[threadIdx.x] = temp;

  __syncthreads();

  // reduction
  unsigned int i = blockDim.x/2;
  while(i != 0){
    if(threadIdx.x < i){
      cache[threadIdx.x] += cache[threadIdx.x + i];
    }
    __syncthreads();
    i /= 2;
  }

  if(threadIdx.x == 0){
    atomicAdd(dot, 0.25*hx_d*hy_d*cache[0]);
  }
  
}


void trapz_wrapper(dim3 grid1D, dim3 block1D, sliceVar slice, sliceVar slice0, 
  stateVar velTan, REAL *integrals, REAL *coeffTrapz,
  int *tip_count, vec5dyn *tip_vector, int count) {

  REAL *prod;
  REAL *prod_d;
  prod = (REAL*)malloc(sizeof(REAL));
  CudaSafeCall(hipMalloc((void**)&prod_d, sizeof(REAL)));

  // It's important to restart the output vector
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  trapz_kernel<<<grid1D, block1D>>>(slice0.ux, slice.ux, slice0.vx, slice.vx,
    prod_d,coeffTrapz,tip_count,tip_vector, count);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(prod, prod_d, sizeof(REAL), hipMemcpyDeviceToHost));
  integrals[0] = *prod;
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  trapz_kernel<<<grid1D, block1D>>>(slice0.ux, slice.uy, slice0.vx, slice.vy,
    prod_d,coeffTrapz,tip_count,tip_vector, count);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(prod, prod_d, sizeof(REAL), hipMemcpyDeviceToHost));
  integrals[1] = *prod;
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  trapz_kernel<<<grid1D, block1D>>>(slice0.ux, slice.ut, slice0.vx, slice.vt, 
    prod_d,coeffTrapz,tip_count,tip_vector, count);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(prod, prod_d, sizeof(REAL), hipMemcpyDeviceToHost));
  integrals[2] = *prod;
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  trapz_kernel<<<grid1D, block1D>>>(slice0.uy, slice.ux, slice0.vy, slice.vx, 
    prod_d,coeffTrapz,tip_count,tip_vector, count);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(prod, prod_d, sizeof(REAL), hipMemcpyDeviceToHost));
  integrals[3] = *prod;
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  trapz_kernel<<<grid1D, block1D>>>(slice0.uy, slice.uy, slice0.vy, slice.vy,
    prod_d,coeffTrapz,tip_count,tip_vector, count);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(prod, prod_d, sizeof(REAL), hipMemcpyDeviceToHost));
  integrals[4] = *prod;
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  trapz_kernel<<<grid1D, block1D>>>(slice0.uy, slice.ut, slice0.vy, slice.vt,
    prod_d,coeffTrapz,tip_count,tip_vector, count);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(prod, prod_d, sizeof(REAL), hipMemcpyDeviceToHost));
  integrals[5] = *prod;
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  trapz_kernel<<<grid1D, block1D>>>(slice0.ut, slice.ux, slice0.vt, slice.vx,
    prod_d,coeffTrapz,tip_count,tip_vector, count);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(prod, prod_d, sizeof(REAL), hipMemcpyDeviceToHost));
  integrals[6] = *prod;
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  trapz_kernel<<<grid1D, block1D>>>(slice0.ut, slice.uy, slice0.vt, slice.vy,
    prod_d,coeffTrapz,tip_count,tip_vector, count);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(prod, prod_d, sizeof(REAL), hipMemcpyDeviceToHost));
  integrals[7] = *prod;
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  trapz_kernel<<<grid1D, block1D>>>(slice0.ut, slice.ut, slice0.vt, slice.vt, 
    prod_d,coeffTrapz,tip_count,tip_vector, count);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(prod, prod_d, sizeof(REAL), hipMemcpyDeviceToHost));
  integrals[8] = *prod;
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  trapz_kernel<<<grid1D, block1D>>>(slice0.ux, velTan.u, slice0.vx, velTan.v,
    prod_d,coeffTrapz,tip_count,tip_vector, count);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(prod, prod_d, sizeof(REAL), hipMemcpyDeviceToHost));
  integrals[9] = *prod;
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  trapz_kernel<<<grid1D, block1D>>>(slice0.uy, velTan.u, slice0.vy, velTan.v,
    prod_d,coeffTrapz,tip_count,tip_vector, count);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(prod, prod_d, sizeof(REAL), hipMemcpyDeviceToHost));
  integrals[10] = *prod;
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  trapz_kernel<<<grid1D, block1D>>>(slice0.ut, velTan.u, slice0.vt, velTan.v,
    prod_d,coeffTrapz,tip_count,tip_vector, count);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(prod, prod_d, sizeof(REAL), hipMemcpyDeviceToHost));
  integrals[11] = *prod;
  CudaSafeCall(hipMemset(prod_d, 0.0, sizeof(REAL)));

  free(prod);
  CudaSafeCall(hipFree(prod_d));

}
