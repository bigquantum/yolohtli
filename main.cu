#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <iostream>

#include <GL/glew.h>
#include <GL/glut.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

#include "typeDefinition.cuh"
// #include "globalVariables.cuh"

// Performance libraries
#include "./common/CudaSafeCall.h"
#include "./common/profile_time.h"

// Function protoypes
#include "hostPrototypes.h"
#include "devicePrototypes.cuh"
#include "openGLPrototypes.h"

using namespace std;

/*------------------------------------------------------------------------
* Decalre global variables
*------------------------------------------------------------------------
*/

size_t pitch;

fileVar strAdress;

paramVar param;

// Constants for device
__constant__ int nx_d, ny_d;
__constant__ REAL dt_d, rx_d, ry_d, hx_d, hy_d, Lx_d, Ly_d, qx4_d, qy4_d, fx4_d, fy4_d;
__constant__ REAL rxy_d, rbx_d, rby_d, rscale_d;
__constant__ REAL invdx_d, invdy_d;
__constant__ REAL tc_d, alpha_d, beta_d, delta_d, eps_d, mu_d, gamma_d, theta_d;
__constant__ REAL boundaryVal_d;
__constant__ bool solidSwitch_d, neumannBC_d, gateDiff_d, anisotropy_d, tipGrad_d;
__constant__ int tipOffsetX_d, tipOffsetY_d;
__constant__ float minVarColor_d, maxVarColor_d;
__constant__ float tipx0_d, tipy0_d;
__constant__ REAL Uth_d, conTh1_d, conTh2_d, conTh3_d;
__constant__ int lap4_d, timeIntOrder_d;

// Kernel setup
dim3 grid0D, block0D, grid1D, block1D, grid2D, block2D;

// Voltage and gate arrays
stateVar gate_h, gateIn_d, gateOut_d, J_d;

// Slices (templates) for the integrals
REAL3 c, c0, phi, phi0;
sliceVar slice, slice0;
stateVar velTan;
REAL *integrals;
std::vector<REAL3> clist;
std::vector<REAL3> philist;

// Advection arrays
advVar advect;
// BFECC (advection integration) arrays
stateVar uf_d, ub_d, ue_d;

// Solid holes //
bool *solid, *solid_d; // For the PDEs
REAL *coeffTrapz, *coeffTrapz_d; // For the integrals
bool *intglArea, *intglArea_d; // For the slices (derivatives)

// Arrays for the tip trajectory
int *tip_count_d;
vec5dyn *tip_vector_d; // Holds the tip trajectory
bool *tip_plot;

// Single cell recordings
std::vector<electrodeVar> electrode;
std::vector<REAL> contourLength;
REAL *point_h, *point_d;
stateVar timeSeries;

// Arrays for the contour tracking
int *contour_count_d;
float3 *contour_vector_d; // Holds the contours
bool *stimArea, *stimArea_d; // Area considered for APD(x,y)
REAL *APD1_d, *APD2_d, *sAPD_d, *dAPD_d, *back_d, *front_d; // sAPD_d: APD spatial field
bool *first_APD_d, *contour_plot; // Auxiliary array. Contour plots
REAL *stimulus, *stimulus_d; // Stimulus area
int sampleIt; // Sample period in iterations per unit time
int stimPeriodIt;
int stimDurationIt;
int stimCount = 0;
bool stimLock = false, stimLockKey = false, stimLockMouse = false;
REAL *timeSeriesContourLength;

// Figures/shapes
float2 *trapzAreaCircle, *stimAreaCircle;

// OpenGL pixel buffer object and texture //
GLuint gl_PBO, gl_Tex;
unsigned int *cmap_rgba; // rgba arrays for plotting
unsigned int *cmap_rgba_data, *plot_rgba_data;
float width, height;
int nsolid, nstep, nsteps, ncol;
GLint window1, window2;
bool timeScreen = true, phaseScreen = false, contourScreen = false;
REAL *timeSeriesCx, *timeSeriesCy, *timeSeriesCt;

// Timer for simulation
int base;
float FPS = 1.0;
int initial_time = time(NULL); // Timer for controlling the fps
int final_time, frame_count; // More fram/fps stuff
int fps;

/*------------------------------------------------------------------------
* Program starts here
*------------------------------------------------------------------------
*/

int main(int argc, char *argv[]) {

if (argc > 1) {
  printf("%d VALUES(s) has been introduced form the EXECUTABLE COMMAND line\n",argc-1);
}

param = startMenu(&strAdress,param,argc,argv);

if ( param.save ) {
  strAdress = saveFileNames(strAdress,&param);
}

/*------------------------------------------------------------------------
* Parameters form command line
*------------------------------------------------------------------------
*/

if (argc > 1) param.stimPeriod = atof(argv[1]);


// Commonly changed variables
param.dt = param.reduceSym ? 0.5*param.dt : param.dt;
param.rx = param.dt*param.Dxx/(param.hx*param.hx);
param.ry = param.dt*param.Dyy/(param.hy*param.hy);
param.rxy = 2.0*param.Dxy*param.dt/(4.0*param.hx*param.hy);
param.qx4 = param.dt*param.Dyy/(param.hy*param.hy*12.0);
param.qy4 = param.dt*param.Dxx/(param.hx*param.hx*12.0);
param.fx4 = param.dt/12.0;
param.fy4 = param.dt/12.0;
sampleIt = round(param.sample/param.dt);
stimPeriodIt = floor(param.stimPeriod/param.dt);
stimDurationIt = floor(param.stimDuration/param.dt);

// Symmetry variables (also for predictor scheme)
c.x = 0.0; c.y = 0.0; c.t = 0.0;
c0.x = 0.0; c0.y = 0.0; c0.t = 0.0;
phi.x = 0.0; phi.y = 0.0; phi.t = 0.0;
phi0.x = 0.0; phi0.y = 0.0; phi0.t = 0.0;

// Kernel setup
grid0D = dim3(1,1,1);
block0D = dim3(1,1,1);
grid1D = dim3(GRIDSIZE_1D,1,1);
block1D = dim3(BLOCKSIZE_1D,1,1);
grid2D = dim3(iDivUp(param.nx,BLOCK_DIM_X),iDivUp(param.ny,BLOCK_DIM_Y),1);
block2D = dim3(BLOCK_DIM_X, BLOCK_DIM_Y,1);

printf("\n**Parameter values**\n");
printf("dt = %f ms \n", param.dt);
printf("rx = %f \n", param.rx);
printf("ry = %f \n", param.ry);
printf("Lx = %f cm \n", param.Lx);
printf("Ly = %f cm \n", param.Ly);
printf("nx = %d \n", param.nx);
printf("ny = %d \n", param.ny);
printf("hx = %f cm \n", param.hx);
printf("hy = %f cm \n", param.hy);
printf("Dx = %f cm^2/ms \n",param.Dxx);
printf("Dy = %f cm^2/ms \n",param.Dyy);

printf("\n");
pressEnterKey();

printf("\n**Keyboard options**\n");
printf("m --> Menu\n");
printf("Esc --> Close application\n");
printf("Space bar --> Pause simulation\n");
printf("r --> Restart simulation\n");
printf("q --> Pace/stimulate\n");
printf("s --> Symmetry reduction\n");
printf("t --> Tip tracjectory recordings\n");
printf("c --> Contour recordings\n");
printf("p --> Print screenshot\n");
printf("/ --> Conduction block\n");
printf("x --> Stimulate all tissue once\n");
printf("x --> Stimulate circle tissue once\n");
printf("z --> Clear screen\n");
printf("1 --> Screen 1 (electrodes)\n");
printf("2 --> Screen 2 (SR velocities)\n");
printf("3 --> Screen 3 (Contour length)\n");

/*------------------------------------------------------------------------
* Array allocation
*------------------------------------------------------------------------
*/

param.memSize = param.nx*param.ny*sizeof(REAL);

// Array allocation
gate_h.u = (REAL*)malloc(param.memSize);
gate_h.v = (REAL*)malloc(param.memSize);

// Holds the results of the slice integrals
integrals = (REAL*)malloc(12*sizeof(REAL));

// Circular boundary
solid = new bool[param.nx*param.ny];
coeffTrapz = (REAL*)malloc(param.memSize);
intglArea = new bool[param.nx*param.ny];

// Stimulation area
stimArea = new bool[param.nx*param.ny];
stimulus = (REAL*)malloc(param.memSize);

// Electrode recordings
point_h = (REAL*)malloc(param.eSize*sizeof(REAL));
timeSeries.u = (REAL*)malloc(param.wnx*sizeof(REAL));
timeSeries.v = (REAL*)malloc(param.wnx*sizeof(REAL));

// Array for figures/shapes
trapzAreaCircle = (float2*)malloc(param.nc*sizeof(float2));
stimAreaCircle = (float2*)malloc(param.nc*sizeof(float2));

// Plotting cx, cy, ct
timeSeriesCx = (REAL*)malloc(param.wnx*sizeof(REAL));
timeSeriesCy = (REAL*)malloc(param.wnx*sizeof(REAL));
timeSeriesCt = (REAL*)malloc(param.wnx*sizeof(REAL));

// Contour length
timeSeriesContourLength = (REAL*)malloc(param.wnx*sizeof(REAL));

// Allocate device memory arrays
CudaSafeCall(hipMalloc(&gateIn_d.u,param.memSize));
CudaSafeCall(hipMalloc(&gateIn_d.v,param.memSize));
CudaSafeCall(hipMalloc(&gateOut_d.u,param.memSize));
CudaSafeCall(hipMalloc(&gateOut_d.v,param.memSize));
CudaSafeCall(hipMalloc(&J_d.u,param.memSize));
CudaSafeCall(hipMalloc(&J_d.v,param.memSize));
CudaSafeCall(hipMalloc(&slice.ux,param.memSize));
CudaSafeCall(hipMalloc(&slice.uy,param.memSize));
CudaSafeCall(hipMalloc(&slice.ut,param.memSize));
CudaSafeCall(hipMalloc(&slice.vx,param.memSize));
CudaSafeCall(hipMalloc(&slice.vy,param.memSize));
CudaSafeCall(hipMalloc(&slice.vt,param.memSize));
CudaSafeCall(hipMalloc(&slice0.ux,param.memSize));
CudaSafeCall(hipMalloc(&slice0.uy,param.memSize));
CudaSafeCall(hipMalloc(&slice0.ut,param.memSize));
CudaSafeCall(hipMalloc(&slice0.vx,param.memSize));
CudaSafeCall(hipMalloc(&slice0.vy,param.memSize));
CudaSafeCall(hipMalloc(&slice0.vt,param.memSize));

CudaSafeCall(hipMalloc(&advect.x,param.memSize));
CudaSafeCall(hipMalloc(&advect.y,param.memSize));
CudaSafeCall(hipMalloc(&velTan.u,param.memSize));
CudaSafeCall(hipMalloc(&velTan.v,param.memSize));

CudaSafeCall(hipMalloc(&tip_plot, param.nx*param.ny*sizeof(bool)));

CudaSafeCall(hipMalloc(&uf_d.u,param.memSize));
CudaSafeCall(hipMalloc(&ub_d.u,param.memSize));
CudaSafeCall(hipMalloc(&ue_d.u,param.memSize));
CudaSafeCall(hipMalloc(&uf_d.v,param.memSize));
CudaSafeCall(hipMalloc(&ub_d.v,param.memSize));
CudaSafeCall(hipMalloc(&ue_d.v,param.memSize));

CudaSafeCall(hipMalloc(&tip_count_d,sizeof(int)));
CudaSafeCall(hipMalloc(&tip_vector_d,TIPVECSIZE*sizeof(vec5dyn)));

CudaSafeCall(hipMalloc(&solid_d, param.nx*param.ny*sizeof(bool)));
CudaSafeCall(hipMalloc(&coeffTrapz_d,param.memSize));
CudaSafeCall(hipMalloc(&intglArea_d,param.nx*param.ny*sizeof(bool)));

CudaSafeCall(hipMalloc(&APD1_d,param.memSize));
CudaSafeCall(hipMalloc(&APD2_d,param.memSize));
CudaSafeCall(hipMalloc(&sAPD_d,param.memSize));
CudaSafeCall(hipMalloc(&dAPD_d,param.memSize));
CudaSafeCall(hipMalloc(&back_d,param.memSize));
CudaSafeCall(hipMalloc(&front_d,param.memSize));
CudaSafeCall(hipMalloc(&first_APD_d, param.nx*param.ny*sizeof(bool)));
CudaSafeCall(hipMalloc(&contour_plot, param.nx*param.ny*sizeof(bool)));
CudaSafeCall(hipMalloc(&stimArea_d, param.nx*param.ny*sizeof(bool)));
CudaSafeCall(hipMalloc(&stimulus_d,param.memSize));
CudaSafeCall(hipMalloc(&contour_count_d,sizeof(int)));
CudaSafeCall(hipMalloc(&contour_vector_d,param.nx*param.ny*sizeof(float3)));

CudaSafeCall(hipMalloc(&point_d,param.eSize*sizeof(REAL)));

/*------------------------------------------------------------------------
* Set GPU constants
*------------------------------------------------------------------------
*/

CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(nx_d), &param.nx, sizeof(int), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ny_d), &param.ny, sizeof(int), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(rx_d), &param.rx, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ry_d), &param.ry, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(qx4_d), &param.qx4, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(qy4_d), &param.qy4, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fx4_d), &param.fx4, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fy4_d), &param.fy4, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(hx_d), &param.hx, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(hy_d), &param.hy, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dt_d), &param.dt, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdx_d), &param.invdx, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdy_d), &param.invdy, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(Lx_d), &param.Lx, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(Ly_d), &param.Ly, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(rxy_d), &param.rxy, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(rbx_d), &param.rbx, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(rby_d), &param.rby, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(rscale_d), &param.rscale, sizeof(REAL), 0,
  hipMemcpyHostToDevice));

CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(boundaryVal_d), &param.boundaryVal, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(solidSwitch_d), &param.solidSwitch, sizeof(bool), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(neumannBC_d), &param.neumannBC, sizeof(bool), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gateDiff_d), &param.gateDiff, sizeof(bool), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(anisotropy_d), &param.anisotropy, sizeof(bool), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(tipGrad_d), &param.tipGrad, sizeof(bool), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lap4_d), &param.lap4, sizeof(int), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(timeIntOrder_d), &param.timeIntOrder, sizeof(int), 0,
  hipMemcpyHostToDevice));

CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(tipOffsetX_d), &param.tipOffsetX, sizeof(int), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(tipOffsetY_d), &param.tipOffsetY, sizeof(int), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(minVarColor_d), &param.minVarColor, sizeof(float), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(maxVarColor_d), &param.maxVarColor, sizeof(float), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(tipx0_d), &param.tipx, sizeof(float), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(tipy0_d), &param.tipy, sizeof(float), 0,
  hipMemcpyHostToDevice));

CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(conTh1_d), &param.contourThresh1, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(conTh2_d), &param.contourThresh2, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(conTh3_d), &param.contourThresh3, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(Uth_d), &param.Uth, sizeof(REAL), 0,
  hipMemcpyHostToDevice));

CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(tc_d), &param.tc, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(alpha_d), &param.alpha, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(beta_d), &param.beta, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gamma_d), &param.gamma, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(delta_d), &param.delta, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(eps_d), &param.eps, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mu_d), &param.mu, sizeof(REAL), 0,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(theta_d), &param.theta, sizeof(REAL), 0,
  hipMemcpyHostToDevice));


printf("Finished allocating device arrays\n");

/*------------------------------------------------------------------------
* Initializing physical arrays. Copy from host to device
*------------------------------------------------------------------------
*/

initGates(pitch,gate_h);

/*------------------------------------------------------------------------
* Initialize device arrays to 0
*------------------------------------------------------------------------
*/

CudaSafeCall(hipMemset(slice.ux, 0.0, param.memSize));
CudaSafeCall(hipMemset(slice.uy, 0.0, param.memSize));
CudaSafeCall(hipMemset(slice.ut, 0.0, param.memSize));
CudaSafeCall(hipMemset(slice.vx, 0.0, param.memSize));
CudaSafeCall(hipMemset(slice.vy, 0.0, param.memSize));
CudaSafeCall(hipMemset(slice.vt, 0.0, param.memSize));
CudaSafeCall(hipMemset(slice0.ux, 0.0, param.memSize));
CudaSafeCall(hipMemset(slice0.uy, 0.0, param.memSize));
CudaSafeCall(hipMemset(slice0.ut, 0.0, param.memSize));
CudaSafeCall(hipMemset(slice0.vx, 0.0, param.memSize));
CudaSafeCall(hipMemset(slice0.vy, 0.0, param.memSize));
CudaSafeCall(hipMemset(slice0.vt, 0.0, param.memSize));
CudaSafeCall(hipMemset(advect.x, 0.0, param.memSize));
CudaSafeCall(hipMemset(advect.y, 0.0, param.memSize));
CudaSafeCall(hipMemset(velTan.u, 0.0, param.memSize));
CudaSafeCall(hipMemset(velTan.v, 0.0, param.memSize));
CudaSafeCall(hipMemset(J_d.u, 0.0, param.memSize));
CudaSafeCall(hipMemset(J_d.v, 0.0, param.memSize));
CudaSafeCall(hipMemset(APD1_d, 0.0f, param.memSize));
CudaSafeCall(hipMemset(APD2_d, 0.0f, param.memSize));

CudaSafeCall(hipMemset(tip_plot,0,param.nx*param.ny*sizeof(bool)));
CudaSafeCall(hipMemset(first_APD_d,0,param.nx*param.ny*sizeof(bool)));
CudaSafeCall(hipMemset(contour_plot,0,param.nx*param.ny*sizeof(bool)));
CudaSafeCall(hipMemset(contour_vector_d,0.0,param.nx*param.ny*sizeof(float3)));
CudaSafeCall(hipMemset(tip_vector_d,0.0,TIPVECSIZE*sizeof(vec5dyn)));

printf("Finished initalizing variables\n");
printf("Starting simulation\n\n");

/*------------------------------------------------------------------------
* Load color RGB
*------------------------------------------------------------------------
*/

loadcmap();

/*------------------------------------------------------------------------
* Create masks for domain, stimulus range and measurements
*------------------------------------------------------------------------
*/

domainObjects(solid,coeffTrapz,intglArea,stimArea,
  stimulus,param.stimMag,argc,argv);

/*------------------------------------------------------------------------
* Copy form host to device
*------------------------------------------------------------------------
*/

// Copy data from host to device
CudaSafeCall(hipMemcpy((void *)gateIn_d.u,(void *)gate_h.u,param.memSize,
  hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpy((void *)gateIn_d.v,(void *)gate_h.v,param.memSize,
  hipMemcpyHostToDevice));

/*------------------------------------------------------------------------
* Copy masks for domain, stimulus range and measurements to device
*------------------------------------------------------------------------
*/

CudaSafeCall(hipMemcpy(solid_d,solid,param.nx*param.ny*sizeof(bool),hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpy(coeffTrapz_d,coeffTrapz,param.memSize,hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpy(intglArea_d,intglArea, param.nx*param.ny*sizeof(bool),hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpy(stimArea_d,stimArea,param.nx*param.ny*sizeof(bool),hipMemcpyHostToDevice));
CudaSafeCall(hipMemcpy(stimulus_d,stimulus, param.memSize, hipMemcpyHostToDevice));

free(solid);
free(coeffTrapz);
free(intglArea);
free(stimArea);
free(stimulus);
free(cmap_rgba);

/*------------------------------------------------------------------------
* Allocating and initializing OpenGL objects
*------------------------------------------------------------------------
*/

CudaSafeCall(hipMemcpy((void *)cmap_rgba_data,
                          (void *)cmap_rgba, sizeof(unsigned int)*ncol,
                          hipMemcpyHostToDevice));

if (false == initGL(&argc, argv)) exit(0);

/*------------------------------------------------------------------------
* Glut loop initialization. Rendering starts here
*------------------------------------------------------------------------
*/

// Return control to the program
glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_GLUTMAINLOOP_RETURNS);
//glutCloseFunc(cleanup);
glutMainLoop();

/*------------------------------------------------------------------------
* Copy from device to host
*------------------------------------------------------------------------
*/

CudaSafeCall(hipMemcpy((void *)gate_h.u,(void *)gateOut_d.u,param.memSize,
  hipMemcpyDeviceToHost));
CudaSafeCall(hipMemcpy((void *)gate_h.v,(void *)gateOut_d.v,param.memSize,
  hipMemcpyDeviceToHost));

/*------------------------------------------------------------------------
* Save data in files
*------------------------------------------------------------------------
*/

if ( param.save && param.savePackage ) {
  saveFile(strAdress,&param,gate_h,electrode,contourLength,param.dt,
    tip_count_d,tip_vector_d,clist,philist,true,contour_count_d,contour_vector_d);
}

/*------------------------------------------------------------------------
* Deallocate all arrays
*------------------------------------------------------------------------
*/

// Free gate host and device memory
free(gate_h.u);
free(gate_h.v);

CudaSafeCall(hipFree(gateIn_d.u));
CudaSafeCall(hipFree(gateIn_d.v));
CudaSafeCall(hipFree(gateOut_d.u));
CudaSafeCall(hipFree(gateOut_d.v));
CudaSafeCall(hipFree(cmap_rgba_data));
CudaSafeCall(hipFree(plot_rgba_data));

CudaSafeCall(hipFree(slice.ux));
CudaSafeCall(hipFree(slice.uy));
CudaSafeCall(hipFree(slice.ut));
CudaSafeCall(hipFree(slice.vx));
CudaSafeCall(hipFree(slice.vy));
CudaSafeCall(hipFree(slice.vt));
CudaSafeCall(hipFree(slice0.ux));
CudaSafeCall(hipFree(slice0.uy));
CudaSafeCall(hipFree(slice0.ut));
CudaSafeCall(hipFree(slice0.vx));
CudaSafeCall(hipFree(slice0.vy));
CudaSafeCall(hipFree(slice0.vt));
CudaSafeCall(hipFree(advect.x));
CudaSafeCall(hipFree(advect.y));
CudaSafeCall(hipFree(velTan.u));
CudaSafeCall(hipFree(velTan.v));
CudaSafeCall(hipFree(J_d.u));
CudaSafeCall(hipFree(J_d.v));

printf("Simulation ended\n");
printf("Physical time: %f ms\n", param.physicalTime);
printf("Real time: %f s\n", param.tiempo);

return 0;

}

void initGates(size_t pitch, stateVar g_h) {

  /*------------------------------------------------------------------------
  * Initialize host arrays
  *------------------------------------------------------------------------
  */

  if ( param.load ) {

    loadData(g_h,strAdress,&param);

  } else {

    int i, j, idx;

    // Array initialization
    memset(g_h.u, 0.0, param.memSize); 
    memset(g_h.v, 0.0, param.memSize);

    // Initial condition
    // for (j=(int)floor(0);j<(int)floor(param.ny);j++) {
    //   for (i=(int)floor(0);i<(int)floor(40);i++) {
    //     idx = i + param.nx*j;
    //     g_h.u[idx] = 1.0;
    //   }
    // }

    // Cross-field stimulation

    for (j=(int)floor(0);j<(int)floor(param.ny);j++) {
      for (i=(int)floor(0);i<(int)floor(param.nx/8);i++) {
        idx = i + param.nx*j;
        g_h.u[idx] = 1.0;
      }
    }

    for (j=(int)floor(param.ny/2);j<(int)floor(param.ny);j++) {
      for (i=(int)floor(0);i<(int)floor(param.nx);i++) {
        idx = i + param.nx*j;
        g_h.v[idx] = 1.0;
      }
    }

  }
}

void domainObjects(bool *solid, REAL *coeffTrapz, bool *intglArea, bool *stimArea,
  REAL *stimulus, REAL stimMag, int argc, char *argv[]) {

  int i, j, idx;

  /*------------------------------------------------------------------------
  * Load unexcitable holes
  * Heterogeneous media
  *------------------------------------------------------------------------
  */

  if ( param.solidSwitch ) {

    float mesh;

    memset(solid, false, param.nx*param.ny*sizeof(bool));

  /*------------------------------------------------------------------------
  * Load domain boundary
  *------------------------------------------------------------------------
  */

    FILE *fp_img;
    char strPath[128];

    if ( param.nx == 512 ) {
      // fp_img = fopen("./common/cBoundary512.dat","r");
      fp_img = fopen("./common/holes512.dat","r");
    }

    if ( param.nx == 800 ) {
      if ( argc>2) {
        sprintf(strPath, "./common/%s",argv[4]);
        fp_img = fopen(strPath,"r");
      } else {
        fp_img = fopen("./common/holes800_1.dat","r");
      }
    }

    if ( param.nx == 1000 ) {
      fp_img = fopen("./common/holes1000.dat","r");
    }

    if ( param.nx == 1024 ) {
      // fp_img = fopen("./common/cBoundary1024.dat","r");
      fp_img = fopen("./common/holes1024.dat","r");
    }

    if ( fp_img==NULL ) {
      printf("Error: can't open circleBoundary file \n");
      exit(0);
    }

    for (i=0;i<param.nx*param.ny;i++){
      fscanf(fp_img, "%f", &mesh);
      // printf("mesh: %d\n", mesh);
      solid[i] = mesh > 0.5 ? true : false;
      // printf("solid %d\n", (int)solid[i]);
    }
    fclose(fp_img);

  }

  /*------------------------------------------------------------------------
  * Slice (derivative) boundary (integration area) 
  *------------------------------------------------------------------------
  */

  // Be sure that the radius of the slice boundary is always smaller than
  // the radius of the domain. If it's not, the derivatives will be 
  // discontinious at the boundary.

  float x0, y0;
  // rdom = 0.5*(Lx-250.0*hx);
  // param.rdomTrapz = 0.5*(param.Lx-(param.tipOffsetX+param.tipOffsetY)*param.hx);
  param.rdomTrapz = 0.5*((param.tipOffsetX+param.tipOffsetY)*param.hx); // Integral area radius

  for (j=0;j<param.ny;j++) {
    for (i=0;i<param.nx;i++) {
      idx = I2D(param.nx,i,j);
      x0 = (float)i*param.hx - 0.5*param.Lx ;
      y0 = (float)j*param.hy - 0.5*param.Ly ;

      intglArea[idx] = ( (x0*x0 + y0*y0) < param.rdomTrapz*param.rdomTrapz ) ? true : false;

    }
  }

  // Points for OpenGL
  for (i=0;i<param.nc;i++) {
    trapzAreaCircle[i].x = (float)( param.nx/2.f+(param.nx-1)/param.Lx*
      param.rdomTrapz*sin((float)i*(2.0*pi)/(param.nc-1)) );
    trapzAreaCircle[i].y = (float)( param.ny/2.f+(param.ny-1)/param.Ly*
      param.rdomTrapz*cos((float)i*(2.0*pi)/(param.nc-1)) );
  }
  

  /*------------------------------------------------------------------------
  * Load trapezoidal coefficients
  *------------------------------------------------------------------------
  */

  REAL *coeffx, *coeffy;
  coeffx = (REAL*)malloc(param.memSize);
  coeffy = (REAL*)malloc(param.memSize);

  memset(coeffx, 0.0, param.memSize);
  memset(coeffy, 0.0, param.memSize);
  memset(coeffTrapz, 0.0, param.nx*param.ny*sizeof(REAL));

  for (i=0;i<param.nx*param.ny;i++) {
    if ( intglArea[i] == true ) {
      coeffx[i] = 2.0;
    }
    if ( intglArea[i] == true && intglArea[i-1] == false ) {
      coeffx[i] = 1.0;
    }
    if ( intglArea[i] == true && intglArea[i+1] == false ) {
      coeffx[i] = 1.0;
    }
  }

  // transpose matrix
  for (j=0;j<param.ny;++j) {
    for (i=0;i<param.nx;++i) {
      coeffy[(i * param.ny) + j] = coeffx[(j * param.nx) + i];
    }
  }

  for (i=0;i<param.nx*param.ny;i++) {
    coeffTrapz[i] = coeffx[i]*coeffy[i]; 
  }

  free(coeffx);
  free(coeffy);

  /*------------------------------------------------------------------------
  * Define the stimulus area
  *------------------------------------------------------------------------
  */

  param.stcx = 0.25*param.Lx; // Stimulus position
  param.stcy = 0.25*param.Ly;
  param.rdomStim = 0.03*param.Lx; // Stimulus radius

  for (j=0;j<param.ny;j++) {
    for (i=0;i<param.nx;i++) {
      idx = I2D(param.nx,i,j);
      x0 = (float)i*param.hx - 0.5*param.Lx ;
      y0 = (float)j*param.hy - 0.5*param.Ly ;

      if ( param.solidSwitch ) {

        // bool sc = solid[idx];
        // bool sw = solid[I2D(param.nx,i-1,j)];
        // bool se = solid[I2D(param.nx,i+1,j)];
        // // Electric field from left to right
        // stimulus[idx] = sc && !sw ?  param.stimMag : 0.0 ; // to the right
        // stimulus[idx] = sc && !se ? -param.stimMag : 0.0 ; // to the left

        // stimulus[idx] = param.stimMag;

        // point stimulus
        stimulus[idx] = ( ((x0-param.stcx)*(x0-param.stcx)+(y0-param.stcy)*(y0-param.stcy)) 
         < param.rdomStim*param.rdomStim ) ? param.stimMag : 0.0;

      } else {

        // For contour stimulation
        // stimulus[idx] = param.stimMag;

        // point stimulus
        stimulus[idx] = ( ((x0-param.stcx)*(x0-param.stcx)+(y0-param.stcy)*(y0-param.stcy)) 
         < param.rdomStim*param.rdomStim ) ? param.stimMag : 0.0;

        // Rectangular stimulus
        // stimulus[idx] = ( (j>20) && (j<30) ) ? param.stimMag : 0.0;
        
      }

    }
  }

  // Point for OpenGL
  param.pointStim.x = param.nx/2+(param.nx-1)/param.Lx*param.stcx;
  param.pointStim.y = param.ny/2+(param.ny-1)/param.Ly*param.stcy;

  /*------------------------------------------------------------------------
  * Define the area arround the stimulus point where we are measureing the APD(x,y) (OPTIONAL)
  *------------------------------------------------------------------------
  */

  for (j=0;j<param.ny;j++) {
    for (i=0;i<param.nx;i++) {
      idx = I2D(param.nx,i,j);
      x0 = (float)i*param.hx - 0.5*param.Lx ;
      y0 = (float)j*param.hy - 0.5*param.Ly ;

      if ( param.solidSwitch ) {

        // bool sc = solid[idx];
        // stimArea[idx] = sc ? true : false;

        // Point stimulus
        stimArea[idx] = ( ((x0-param.stcx)*(x0-param.stcx)+(y0-param.stcy)*(y0-param.stcy)) 
         < param.rdomAPD*param.rdomAPD ) ? false : true;

      } else {

        // stimArea[idx] = true;

      // Point stimulus
      // stimArea[idx] = ( ((x0-param.stcx)*(x0-param.stcx)+(y0-param.stcy)*(y0-param.stcy)) 
      //  < param.rdomAPD*param.rdomAPD ) ? false : true;

      // Rectangular stimulus
      stimArea[idx] = (j<35) ? false : true;

      }

    }
  }

  // Points for OpenGL
  for (i=0;i<param.nc;i++) {
    stimAreaCircle[i].x = param.nx/2+(param.nx-1)/param.Lx*param.stcx+(param.nx-1)/param.Lx*
      param.rdomAPD*sin((float)i*(2.0*pi)/(param.nc-1));
    stimAreaCircle[i].y = param.ny/2+(param.ny-1)/param.Ly*param.stcy+(param.nx-1)/param.Ly*
      param.rdomAPD*cos((float)i*(2.0*pi)/(param.nc-1));
  }

}

/*------------------------------------------------------------------------
* Here is where all the kernel calls and tip tracking, etc. are made
*------------------------------------------------------------------------
*/

// This function is called automatically, over and over again,  by GLUT
void display(void) {

  glutSetWindow(window1);

  if (param.animate) {

    #pragma unroll
    for (int i=0;i<(param.itPerFrame);i++) {

      /*------------------------------------------------------------------------
      * Reaction-diffusion Solver
      *------------------------------------------------------------------------
      */

      if ( !param.reduceSym ) {

        /* Reaction diffusion */
        reactionDiffusion_wrapper(pitch,grid2D,block2D,gateOut_d,gateIn_d,J_d,
          velTan,param.reduceSym,solid_d,stimLock,stimulus_d,stimLockMouse,param.point);

        swapSoA(&gateIn_d,&gateOut_d);

        param.count++;
        param.physicalTime = param.dt*(REAL)param.count;

      } 

      /*------------------------------------------------------------------------
      * Symmetry reduction Solver
      *------------------------------------------------------------------------
      */

      if ( param.reduceSym ) {

        reactionDiffusion_wrapper(pitch,grid2D,block2D,gateOut_d,gateIn_d,J_d,
          velTan,param.reduceSym,solid_d,stimLock,stimulus_d,stimLockMouse,param.point);

        // if ( (param.count%sampleIt == 0) ) {
          tip_wrapper(pitch,grid2D,block2D,gateIn_d,gateOut_d,velTan,param.physicalTime,
            param.tipAlgorithm,param.recordTip,tip_plot,tip_count_d,tip_vector_d);
          clist.push_back(c);
          philist.push_back(phi);
        // }

        slice_wrapper(pitch,grid2D,block2D,gateIn_d,
          slice,slice0,param.reduceSym,param.reduceSymStart,advect,2,intglArea_d,
          tip_count_d,tip_vector_d,param.count);

        if ( param.count == 0 ) {
          trapz_wrapper(grid1D,block2D,slice,slice0,velTan,integrals,coeffTrapz_d,
            tip_count_d,tip_vector_d,param.count);

          c = solve_matrix(c,phi,integrals);

          Cxy_field_wrapper(pitch,grid2D,block2D,advect,c,phi,solid_d);

          slice_wrapper(pitch,grid2D,block2D,gateIn_d,
            slice,slice0,param.reduceSym,param.reduceSymStart,advect,2,intglArea_d,
            tip_count_d,tip_vector_d,param.count);
        }

        trapz_wrapper(grid1D,block1D,slice,slice0,velTan,integrals,coeffTrapz_d
          ,tip_count_d,tip_vector_d,param.count);

        c = solve_matrix(c,phi,integrals);
        // c.y = integrals[10]/integrals[4]; // Translational symmetry in y
        // c.x = integrals[9]/integrals[0]; // Translational symmetry in x

        Cxy_field_wrapper(pitch,grid2D,block2D,advect,c,phi,solid_d);

        advFDBFECC_wrapper(pitch,grid2D,block2D,gateIn_d,gateOut_d,advect,
          uf_d,ub_d,ue_d,solid_d);

        if ( param.reduceSymStart ) {
          phi.x = phi.x+c.x*param.dt;
          phi.y = phi.y+c.y*param.dt;
          phi.t = phi.t+c.t*param.dt;
          c0.x = c.x;
          c0.y = c.y;
          c0.t = c.t;
        } else {
          phi.x = phi.x+param.dt*(1.5*c.x-0.5*c0.x);
          phi.y = phi.y+param.dt*(1.5*c.y-0.5*c0.y);
          phi.t = phi.t+param.dt*(1.5*c.t-0.5*c0.t);
          c0.x = c.x;
          c0.y = c.y;
          c0.t = c.t;
        }

        param.count++;
        param.physicalTime = param.dt*(REAL)param.count;

      }

      /*------------------------------------------------------------------------
      * Special functions
      *------------------------------------------------------------------------
      */

      if ( param.count%sampleIt == 0 ) {
        if ( param.plotTip && !param.reduceSym ) {
          tip_wrapper(pitch,grid2D,block2D,gateIn_d,gateOut_d,velTan,param.physicalTime,
            param.tipAlgorithm,param.recordTip,tip_plot,tip_count_d,tip_vector_d);
        }
        if ( param.recordTip ) {
          saveFile(strAdress,&param,gate_h,electrode,contourLength,param.dt,tip_count_d,tip_vector_d, 
            clist,philist,false,contour_count_d,contour_vector_d);
        }
        if ( param.plotContour ) {
          if ( param.contourMode==1 ) {
            countour_wrapper(pitch,grid2D,block2D,gateOut_d.u,sAPD_d,contour_plot,stimArea_d,
              contour_count_d,contour_vector_d,param.physicalTime,param.contourMode);
          }
          if ( ( param.contourMode==2 ) || ( param.contourMode==3 ) ) {
            countour_wrapper(pitch,grid2D,block2D,gateOut_d.u,gateOut_d.v,contour_plot,stimArea_d,
              contour_count_d,contour_vector_d,param.physicalTime,param.contourMode);
          }
          if ( param.recordContour ) {
            saveFile(strAdress,&param,gate_h,electrode,contourLength,param.dt,tip_count_d,tip_vector_d,
              clist,philist,false,contour_count_d,contour_vector_d);
          }
        }
        if ( param.saveEveryIt && (param.physicalTime>param.startRecTime) ) {
          CudaSafeCall(hipMemcpy((void *)gate_h.u,(void *)gateOut_d.u,param.memSize,
            hipMemcpyDeviceToHost));
          saveFile(strAdress,&param,gate_h,electrode,contourLength,param.dt,tip_count_d,tip_vector_d, 
            clist,philist,false,contour_count_d,contour_vector_d);
        }
      }

      // Automatic periodic pacing
      if ( param.stimulate ) {
        if ( (param.count%stimPeriodIt == 0) || (stimCount > 0) ) {
          stimLock = true;
          stimCount++;
          if ( stimCount>stimDurationIt ) {
          	param.leapShocks++;
            stimLock = false;
            stimCount = 0;
          }
        }
      }

      // // Find if the tissue is still excited tissue
      // if ( !isThereFib(timeSeries.u,param) ) {
      //   param.fibTerminated = true;
      //   glutLeaveMainLoop();
      //   glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE,GLUT_ACTION_GLUTMAINLOOP_RETURNS);
      //   printf("No fibrillation. Abort program\n");
      // }

      // Stim with keyboard
      if ( stimLockKey ) {
        stimLock = true;
        stimCount++;
        if ( stimCount>stimDurationIt ) {
          stimLock = false;
          stimCount = 0;
          stimLockKey = false;
        }
      }

      // Stim with mouse
      if ( stimLockMouse ) {
        stimLock = true;
        stimCount++;
        if ( stimCount>stimDurationIt ) {
          stimLock = false;
          stimCount = 0;
          stimLockMouse = false;
        }
      }

      if ( param.contourMode==1 ) {
        sAPD_wrapper(pitch,grid1D,block1D,param.count,gateIn_d.u,gateOut_d.u,APD1_d,APD2_d,sAPD_d,dAPD_d,
          back_d,front_d,first_APD_d,stimArea_d,param.stimulate);
      }

      singleCell_wrapper(pitch,grid0D,block0D,gateOut_d,param.eSize,point_h,point_d,
        param.point);

    } // ITPERFRAME for loop ends

    /*------------------------------------------------------------------------
    * Plot in screen
    *------------------------------------------------------------------------
    */

    // For plotting, map the plot_rgba_data array to the
    // gl_PBO pixel buffer
    CudaSafeCall(cudaGLMapBufferObject((void**)&plot_rgba_data, gl_PBO));

    if ( param.plotTip ) {
      get_rgba_wrapper(pitch,grid2D,block2D,ncol,gateOut_d.u,plot_rgba_data,cmap_rgba_data,tip_plot);
    } else if ( param.plotContour ) {
      get_rgba_wrapper(pitch,grid2D,block2D,ncol,gateOut_d.u,plot_rgba_data,cmap_rgba_data,contour_plot);
    } else {
      get_rgba_wrapper(pitch,grid2D,block2D,ncol,gateOut_d.u,plot_rgba_data,cmap_rgba_data,tip_plot);
    }

    /*------------------------------------------------------------------------
    * Update time series
    *------------------------------------------------------------------------
    */

    int *contour_pts;
    contour_pts = (int*)malloc(sizeof(int));
    CudaSafeCall(hipMemcpy(contour_pts,contour_count_d,sizeof(int),hipMemcpyDeviceToHost));

    if ( param.plotTimeSeries ) {

      // Voltage, gates
      #pragma unroll
      for (int i=0;i<(param.wnx-1);i++) {
        timeSeries.u[i] = timeSeries.u[i+1]; // Shift elements to the left
        timeSeries.v[i] = timeSeries.v[i+1];

        // Symmetry reduction
        if ( param.reduceSym ) {
          timeSeriesCx[i] = timeSeriesCx[i+1];
          timeSeriesCy[i] = timeSeriesCy[i+1];
          timeSeriesCt[i] = timeSeriesCt[i+1];
        }
        timeSeriesContourLength[i] = timeSeriesContourLength[i+1];
      }

      // Voltage, gates
      timeSeries.u[param.wnx-1] = point_h[0];
      timeSeries.v[param.wnx-1] = point_h[1];
      // Symmetry reduction
      if ( param.reduceSym ) {
        timeSeriesCx[param.wnx-1] = c.x;
        timeSeriesCy[param.wnx-1] = c.y;
        timeSeriesCt[param.wnx-1] = c.t;
      }

      // Contours
      timeSeriesContourLength[param.wnx-1] = (REAL)*contour_pts;
    }

    if ( param.recordTimeSeries ) {
      if ( param.count%sampleIt == 0 ) {

        // Record single-cell electrodes
        electrodeVar data = {
          point_h[0],
          point_h[1],
        };
        electrode.push_back(data);

        // Record contour length
        contourLength.push_back((REAL)*contour_pts);

      }
    }

    free(contour_pts);

  }

  /*------------------------------------------------------------------------
  * Time limit. Terminate simulation
  *------------------------------------------------------------------------
  */

  if ( param.physicalTime >= (param.physicalTimeLim-1.0) ) {
    glutLeaveMainLoop();
    glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE,GLUT_ACTION_GLUTMAINLOOP_RETURNS);
  }
  
  CudaSafeCall(cudaGLUnmapBufferObject(gl_PBO));

  // Copy the pixel buffer to the texture, ready to display
  glTexSubImage2D(GL_TEXTURE_2D,0,0,0,param.nx,param.ny,GL_RGBA,GL_UNSIGNED_BYTE,0);

  // Render one quad to the screen and colour it using our texture
  // i.e. plot our plotvar data to the screen
  glClear(GL_COLOR_BUFFER_BIT);
  glBegin(GL_QUADS);
  glTexCoord2f (0.0, 0.0);
  glVertex3f (0.0, 0.0, 0.0);
  glTexCoord2f (1.0, 0.0);
  glVertex3f (param.nx, 0.0, 0.0);
  glTexCoord2f (1.0, 1.0);
  glVertex3f (param.nx, param.ny, 0.0);
  glTexCoord2f (0.0, 1.0);
  glVertex3f (0.0, param.ny, 0.0);
  glEnd();

  /*------------------------------------------------------------------------
  * Add figures to screen
  *------------------------------------------------------------------------
  */

  addFigures(param.point,param.pointStim,trapzAreaCircle,stimAreaCircle,
    param,tip_count_d,tip_vector_d);

  glutSwapBuffers();

  computeFPS();

}

void mouse(int button, int state, int x, int y) {

// GLUT mouse callback. Left button draws the solid, right button removes solid

  float xx,yy;

  if ((button == GLUT_LEFT_BUTTON) && (state == GLUT_DOWN)) {
    xx = x;
    yy = y;
    param.point.x = xx/width*param.nx;
    param.point.y = (height-yy)/height*param.ny;

  }

  // glutPostRedisplay();
}

void displaySingleCell(void) {

  glutSetWindow(window2);
  glutPostRedisplay();
  glClear(GL_COLOR_BUFFER_BIT);
  glLoadIdentity();

  /*------------------------------------------------------------------------
  * Time series plot
  *------------------------------------------------------------------------
  */

  if (timeScreen) {

    glTranslatef(0.0,0.1*param.wnx,0.0);

    //////////////////// Grid

    glLineWidth(4.0);
    glColor3f(0.752,0.752, 0.752); // gray
    glPushAttrib(GL_ENABLE_BIT); 
    glLineStipple(1, 0x0F0F);
    glEnable(GL_LINE_STIPPLE);
    glBegin(GL_LINES);
    // Horizontal lines
    glVertex2f(0.0, 0.8*param.wny);
    glVertex2f(param.wnx, 0.8*param.wny);
    glVertex2f(0.0, 0.6*param.wny);
    glVertex2f(param.wnx, 0.6*param.wny);
    glVertex2f(0.0, 0.4*param.wny);
    glVertex2f(param.wnx, 0.4*param.wny);
    glVertex2f(0.0, 0.2*param.wny);
    glVertex2f(param.wnx, 0.2*param.wny);
    glVertex2f(0.0, 0.0);
    glVertex2f(param.wnx, 0.0);
    // Ticks
    glVertex2f(0.2*param.wnx,-10.0);
    glVertex2f(0.2*param.wnx,0.8*param.wny+10.0);
    glVertex2f(0.4*param.wnx,-10.0);
    glVertex2f(0.4*param.wnx,0.8*param.wny+10.0);
    glVertex2f(0.6*param.wnx,-10.0);
    glVertex2f(0.6*param.wnx,0.8*param.wny+10.0);
    glVertex2f(0.8*param.wnx,-10.0);
    glVertex2f(0.8*param.wnx,0.8*param.wny+10.0);

    glEnd();
    glPopAttrib();

    //////////////////// voltage time series

    glLineWidth(2.0);
    glColor3f(0.0,0.0,1.0); // blue
    glEnable(GL_LINE_SMOOTH);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glBegin(GL_LINE_STRIP);

    #pragma unroll
    for (int i=0;i<param.wnx;i++) {
      float t = (float)i;
      float y = 0.8*param.wny*timeSeries.u[i] / abs(param.maxVarColor-param.minVarColor);
      glVertex2d(t,y);
    }
    glEnd();

    glLineWidth(2.0);
    glColor3f(1.0,0.0,0.0); // red
    glEnable(GL_LINE_SMOOTH);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glBegin(GL_LINE_STRIP);

    #pragma unroll
    for (int i=0;i<param.wnx;i++) {
      float t = (float)i;
      float y = 0.8*param.wny*timeSeries.v[i] / abs(param.maxVarColor-param.minVarColor);
      glVertex2d(t,y);
    }
    glEnd();

  }

  /*------------------------------------------------------------------------
  * Phase space plot
  *------------------------------------------------------------------------
  */

  if ( phaseScreen && param.reduceSym ) {

    float cScale = 2.0;

    glTranslatef(0.0,0.5*param.wnx,0.0);

    //////////////////// Grid

    glLineWidth(4.0);
    glColor3f(0.752,0.752, 0.752); // gray
    glPushAttrib(GL_ENABLE_BIT); 
    glLineStipple(1, 0x0F0F);
    glEnable(GL_LINE_STIPPLE);
    glBegin(GL_LINES);

    // Horizontal lines
    glVertex2f(0.0, 0.0);
    glVertex2f(param.wnx, 0.0);
    glVertex2f(0.0, 0.25*param.wny);
    glVertex2f(param.wnx, 0.25*param.wny);
    glVertex2f(0.0, -0.25*param.wny);
    glVertex2f(param.wnx, -0.25*param.wny);

    // Vertical lines
    glVertex2f(0.2*param.wnx,-0.5*param.wnx);
    glVertex2f(0.2*param.wnx,0.5*param.wnx);
    glVertex2f(0.4*param.wnx,-0.5*param.wnx);
    glVertex2f(0.4*param.wnx,0.5*param.wnx);
    glVertex2f(0.6*param.wnx,-0.5*param.wnx);
    glVertex2f(0.6*param.wnx,0.5*param.wnx);
    glVertex2f(0.8*param.wnx,-0.5*param.wnx);
    glVertex2f(0.8*param.wnx,0.5*param.wnx);

    glEnd();
    glPopAttrib();

    //////////////////// Velocity (symmetry reduction) time series

    glLineWidth(2.0);
    glColor3f(0.0,0.0,1.0); // blue
    glEnable(GL_LINE_SMOOTH);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glBegin(GL_LINE_STRIP);

    #pragma unroll
    for (int i=0;i<param.wnx;i++) {
      float t = (float)i;
      float y = param.wny*timeSeriesCx[i]*cScale;
      glVertex2d(t,y);
    }
    glEnd();

    glLineWidth(2.0);
    glColor3f(1.0,0.0,0.0); // red
    glEnable(GL_LINE_SMOOTH);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glBegin(GL_LINE_STRIP);

    #pragma unroll
    for (int i=0;i<param.wnx;i++) {
      float t = (float)i;
      float y = param.wny*timeSeriesCy[i]*cScale;
      glVertex2d(t,y);
    }
    glEnd();

    glLineWidth(2.0);
    glColor3f(0.0,1.0,0.0); // green
    glEnable(GL_LINE_SMOOTH);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glBegin(GL_LINE_STRIP);

    #pragma unroll
    for (int i=0;i<param.wnx;i++) {
      float t = (float)i;
      float y = param.wny*timeSeriesCt[i]*cScale;
      glVertex2d(t,y);
    }
    glEnd();

  }

  if ( contourScreen && param.plotContour ) {

    float cScale = 1.0/param.ny*0.05;

    glTranslatef(0.0,0.1*param.wnx,0.0);

    //////////////////// Grid

    glLineWidth(4.0);
    glColor3f(0.752,0.752, 0.752); // gray
    glPushAttrib(GL_ENABLE_BIT); 
    glLineStipple(1, 0x0F0F);
    glEnable(GL_LINE_STIPPLE);
    glBegin(GL_LINES);
    // Horizontal lines
    glVertex2f(0.0, 0.8*param.wny);
    glVertex2f(param.wnx, 0.8*param.wny);
    glVertex2f(0.0, 0.6*param.wny);
    glVertex2f(param.wnx, 0.6*param.wny);
    glVertex2f(0.0, 0.4*param.wny);
    glVertex2f(param.wnx, 0.4*param.wny);
    glVertex2f(0.0, 0.2*param.wny);
    glVertex2f(param.wnx, 0.2*param.wny);
    glVertex2f(0.0, 0.0);
    glVertex2f(param.wnx, 0.0);
    // Ticks
    glVertex2f(0.2*param.wnx,-10.0);
    glVertex2f(0.2*param.wnx,0.8*param.wny+10.0);
    glVertex2f(0.4*param.wnx,-10.0);
    glVertex2f(0.4*param.wnx,0.8*param.wny+10.0);
    glVertex2f(0.6*param.wnx,-10.0);
    glVertex2f(0.6*param.wnx,0.8*param.wny+10.0);
    glVertex2f(0.8*param.wnx,-10.0);
    glVertex2f(0.8*param.wnx,0.8*param.wny+10.0);

    glEnd();
    glPopAttrib();

    //////////////////// Contour length series

    glLineWidth(2.0);
    glColor3f(0.0,0.0,1.0); // blue
    glEnable(GL_LINE_SMOOTH);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glBegin(GL_LINE_STRIP);

    #pragma unroll
    for (int i=0;i<param.wnx;i++) {
      float t = (float)i;
      float y = 0.8*param.wny*timeSeriesContourLength[i]*cScale;
      glVertex2d(t,y);
    }
    glEnd();

  }

  glutSwapBuffers();


}


void resize(int w, int h) {

  // GLUT resize callback to allow us to change the window size

  width = w;
  height = h;
  // glViewport (0, 0,w-param.nx, h);
  glViewport (0, 0, w, h);
  glMatrixMode (GL_PROJECTION);
  glLoadIdentity ();
  glOrtho (0., param.nx, 0., param.ny, -200. ,200.);
  glMatrixMode (GL_MODELVIEW);
  glLoadIdentity ();


}

void loadcmap(void) {

  /*------------------------------------------------------------------------
  * Load RGB colors
  *------------------------------------------------------------------------
  */

  int i;
  float rcol,gcol,bcol;
  FILE *fp_col;

  //
  // Read in colourmap data for OpenGL display
  //
  fp_col = fopen("./common/yolitzincmap.dat","r");

  if (fp_col==NULL) {
    printf("Error: can't open cmap.dat \n");
    exit(0);
  }

  fscanf (fp_col, "%d", &ncol);
  cmap_rgba = (unsigned int *)malloc(ncol*sizeof(unsigned int));
  CudaSafeCall(hipMalloc((void **)&cmap_rgba_data, sizeof(unsigned int)*ncol));

  for (i=0;i<ncol;i++) {
    fscanf(fp_col, "%f%f%f", &rcol, &gcol, &bcol);
    cmap_rgba[i]=((int)(255.0f) << 24) | // convert colourmap to int
    ((int)(bcol * 255.0f) << 16) |
    ((int)(gcol * 255.0f) <<  8) |
    ((int)(rcol * 255.0f) <<  0);
  }

  fclose(fp_col);

}

/*------------------------------------------------------------------------
* Graphics stuff
*------------------------------------------------------------------------
*/

int initGL(int *argc, char **argv) {

  //
  // Iinitialise OpenGL display - use glut
  //
  glutInit(argc, argv);

  /*------------------------------------------------------------------------
  * Spiral wave window
  *------------------------------------------------------------------------
  */

  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
  glutInitWindowSize(param.nx, param.ny);                   // Window of nx x ny pixels
  glutInitWindowPosition(800, 50);               // Window position

  window1 = glutCreateWindow("2V-Voltage");         // Window title

  printf("Loading extensions: %s\n", glewGetErrorString(glewInit()));
  if(!glewIsSupported(
                      "GL_VERSION_2_0 "
                      "GL_ARB_pixel_buffer_object "
                      "GL_EXT_framebuffer_object "
                      )){
      fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
      fflush(stderr);
      return 1;
  }

  // Set up view
  glClearColor(0.0, 0.0, 0.0, 0.0);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glOrtho(0,param.nx,0.,param.ny, -200.0, 200.0);

  // Create texture and bind to gl_Tex
  glEnable(GL_TEXTURE_2D);
  glGenTextures(1, &gl_Tex);                     // Generate 2D texture
  glBindTexture(GL_TEXTURE_2D, gl_Tex);          // bind to gl_Tex
  // texture properties:
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, param.nx, param.ny, 0,
               GL_RGBA, GL_UNSIGNED_BYTE, NULL);
  printf("Texture created.\n");

  // Create pixel buffer object and bind to gl_PBO
  glGenBuffers(1, &gl_PBO);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_PBO);
  unsigned int sizeGL = param.nx*param.ny*sizeof(float); // pitch*param.ny
  glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, sizeGL, NULL, GL_STREAM_COPY);

  CudaSafeCall( cudaGLRegisterBufferObject(gl_PBO));
  printf("Buffer created.\n");

  glutKeyboardFunc(keyboard);
  glutMouseFunc(mouse);
  glutReshapeFunc(resize);
  glutIdleFunc(display);
  glutTimerFunc(1000/FPS,idle,0); // Timer function will be called after 1000/FPS

  /*------------------------------------------------------------------------
  * Time series window
  *------------------------------------------------------------------------
  */

  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
  glutInitWindowSize(param.wnx, param.wny);                   // Window of nx x ny pixels
  glutInitWindowPosition(800+param.nx+10, 50);               // Window position

  window2 = glutCreateWindow("Time series");         // Window title

  printf("Loading extensions: %s\n", glewGetErrorString(glewInit()));
  if(!glewIsSupported(
                      "GL_VERSION_2_0 "
                      "GL_ARB_pixel_buffer_object "
                      "GL_EXT_framebuffer_object "
                      )){
      fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
      fflush(stderr);
      return 1;
  }

  // Set up view
  glClearColor(1.0,1.0,1.0,0.0);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glOrtho(0,param.wnx,0.,param.wny, -200.0, 200.0);

  printf("Starting GLUT main loop...\n\n");

  glutKeyboardFunc(keyboard);
  glutReshapeFunc(resize);
  glutDisplayFunc(displaySingleCell);
  // Subwin = glutCreateSubWindow(mainWindow, param.nx, 0, param.nx, param.ny);

  return true;

}

void idle(int) {

  glutPostRedisplay();
  glutTimerFunc(1000/FPS,idle,0); // Timer function will be called after 1000/FPS

}

void cleanup(void) {

  /*------------------------------------------------------------------------
  * Clean, deallocate instructions for after the simulation has ended
  *------------------------------------------------------------------------
  */

  // Free gate host and device memory
  free(gate_h.u); free(gate_h.v);
  CudaSafeCall(hipFree(gateIn_d.u)); CudaSafeCall(hipFree(gateIn_d.v));
  CudaSafeCall(hipFree(gateOut_d.u)); CudaSafeCall(hipFree(gateOut_d.v));

  puts("\nSimulation ended\n");

  //glBindBuffer(GL_ARRAY_BUFFER, 0);
  //glDeleteBuffers(1, &vbo);

}

__global__ void get_rgba_kernel (size_t pitch, int ncol,
                                 REAL *field,
                                 unsigned int *plot_rgba_data,
                                 unsigned int *cmap_rgba_data,
                                 bool *lines) {

  /*------------------------------------------------------------------------
  * CUDA kernel to fill plot_rgba_data array for plotting
  *------------------------------------------------------------------------
  */

  int icol;
  REAL frac;

  const int i = blockIdx.x*BLOCK_DIM_X + threadIdx.x;
  const int j = blockIdx.y*BLOCK_DIM_Y + threadIdx.y;

  if ( (i<nx_d) && (j<ny_d) ) {

    const int i2d = i + j*nx_d;
    // Change the member of plot_data. to plot a different variable
    frac = (field[i2d]-minVarColor_d)/(maxVarColor_d-minVarColor_d);
    icol = (int)((float)frac*(float)ncol);
    plot_rgba_data[i2d] = (unsigned int)(!lines[i2d]) * cmap_rgba_data[icol];

  }

}

void get_rgba_wrapper(size_t pitch, dim3 grid2D, dim3 block2D, int ncol,
  REAL *field, unsigned int *plot_rgba_data, unsigned int *cmap_rgba_data,
   bool *lines) {

  get_rgba_kernel<<<grid2D,block2D>>>(pitch, ncol,
  	field, plot_rgba_data, cmap_rgba_data, lines); // gateIn_d.u, tip_plot contour_plot
  CudaCheckError();

}

void keyboard(unsigned char key, int x, int y) {

  switch (key) {

    case 'm':
      printf("\n**Keyboard options**\n");
      printf("m --> Menu\n");
      printf("Esc --> Close application\n");
      printf("Space bar --> Pause simulation\n");
      printf("r --> Restart simulation\n");
      printf("q --> Pace/stimulate\n");
      printf("s --> Symmetry reduction\n");
      printf("t --> Tip tracjectory recordings\n");
      printf("c --> Contour recordings\n");
      printf("p --> Print screenshot\n");
      printf("/ --> Conduction block\n");
      printf("x --> Stimulate all tissue once\n");
      printf("x --> Stimulate circle tissue once\n");
      printf("z --> Clear screen\n");
      printf("1 --> Screen 1 (electrodes)\n");
      printf("2 --> Screen 2 (SR velocities)\n");
      printf("3 --> Screen 3 (Contour length)\n");

    break;

    case 27:
      //glutCloseFunc(cleanup);
      // Exit glutMainLoop()
      glutLeaveMainLoop();
      //glutDestroyWindow(glutGetWindow());
      // Return control to the program
      glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE,GLUT_ACTION_GLUTMAINLOOP_RETURNS);
      printf("Abort program\n");
    break;

    case ' ':
        param.animate = !param.animate;
      if ( param.animate ) {
        printf("unPause\n");
      } else {
        printf("Pause\n");
      }
    break;

    case 'q':
      param.stimulate = !param.stimulate;
      if (param.stimulate) {
        printf("Pacing stimulus\n");
      } else {
        printf("No pacing stimulus\n");
      }
    break;

    case 's':
      printf("clist (symmetry reduction) size %lu\n",clist.size());
      printf("philist (symmetry reduction) size %lu\n",philist.size());
      // Clear memory
      while (!clist.empty()) { 
        clist.pop_back();
      }
      // Clear memory
      while (!philist.empty()) { 
        philist.pop_back();
      }
      param.reduceSym = !param.reduceSym;
      param.reduceSymStart = param.reduceSym;
      if (param.reduceSym) {
        param.dt = 0.5*param.dt;
        param.plotTip = true;
        printf("Symmetry reduction activated\n");
      } else {
        param.dt = 2.0*param.dt;
        param.plotTip = false;
        printf("No symmetry reduction\n");
      }

      if (param.plotTip) {
        printf("Plotting tip trajectory\n");
      } else {
        printf("NOT plotting tip trajectory\n");
      }
    break;

    case 't':
      CudaSafeCall(hipMemset(tip_count_d,0,sizeof(int))); // Initialize number of contour points
      int *tip_pts;
      tip_pts = (int*)malloc(sizeof(int));
      CudaSafeCall(hipMemcpy(tip_pts,tip_count_d,sizeof(int),hipMemcpyDeviceToHost));
      CudaSafeCall(hipMemset(tip_vector_d,0,(*tip_pts)*sizeof(vec5dyn)));
      free(tip_pts);
      param.plotTip = !param.plotTip;
      if (param.plotTip) {
        printf("Plotting tip trajectory\n");
      } else {
        printf("NOT recording tip trajectory\n");
      }
    break;

    case 'c':
      CudaSafeCall(hipMemset(contour_count_d,0,sizeof(int))); // Initialize number of contour points
      int *contour_pts;
      contour_pts = (int*)malloc(sizeof(int));
      CudaSafeCall(hipMemcpy(contour_pts,contour_count_d,sizeof(int),hipMemcpyDeviceToHost));
      CudaSafeCall(hipMemset(contour_vector_d,0,(*contour_pts)*sizeof(float3)));
      free(contour_pts);
      param.plotContour = !param.plotContour;
      if (param.plotContour) {
        printf("Recording contours\n");
      } else {
        printf("NOT recording contours\n");
      }
    break;

    case 'r':
      initGates(pitch,gate_h);
      // Copy data from host to device
      CudaSafeCall(hipMemcpy((void *)gateIn_d.u,(void *)gate_h.u,param.memSize,
        hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy((void *)gateIn_d.v,(void *)gate_h.v,param.memSize,
        hipMemcpyHostToDevice));
      printf("Reset to initial condition\n");
    break;

    case 'p':
      screenShot(param.nx, param.ny);
      printf("Screenshot taken\n");
    break;

    case '/':
      conductionBlock(param.memSize,param.counterclock,param.clock,gate_h,gateIn_d);
      printf("Conduction block applied\n");
    break;

    case 'x':
      stimLockKey = true;
      printf("Stimulating (keyboard) once of %2.1f\n", param.stimMag);
    break;

    case 'o':
      stimLockMouse = true;
	  printf("Stimulating (mouse) once of %2.1f\n", param.stimMag);
	break;

    case 'z':
      CudaSafeCall(hipMemset(tip_plot,0,param.nx*param.ny*sizeof(bool)));
      CudaSafeCall(hipMemset(contour_plot,0,param.nx*param.ny*sizeof(bool)));
      printf("Clear screen\n");
    break;

    case '1':
      timeScreen = true;
      phaseScreen = false;
      contourScreen = false;
    break;

    case '2':
      timeScreen = false;
      phaseScreen = true;
      contourScreen = false;
    break;

    case '3':
      timeScreen = false;
      phaseScreen = false;
      contourScreen = true;
    break;

    default:
      puts("No function assigned to this key");
    break;

    }
}

void computeFPS(void) {

  // Count frames per second
  frame_count++;
  final_time = time(NULL);
  if ( (final_time - initial_time) > 0) {
    // frames drawn / time taken (seconds)
    fps = frame_count / (final_time-initial_time);
    // printf("FPS : %d\n", frame_count / (final_time-initial_time));
    frame_count = 0;
    initial_time = final_time;
  }

  GLint64 timer;
  glGetInteger64v(GL_TIMESTAMP, &timer);

  if (param.firstFPS) {
    param.tiempo = 0.0f;
	  base = timer*0.000000001;
	  param.tiempo = timer*0.000000001-base;
    param.firstFPS = false;
    return;
  }

  param.tiempo = timer*0.000000001-base;

  char windowName[256];
  sprintf(windowName, "PhyT %.0f ms | ExcT %.1f s | FPS %d",
    param.physicalTime, param.tiempo, fps);
  glutSetWindowTitle(windowName);
}
