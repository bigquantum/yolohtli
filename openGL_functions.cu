#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <GL/glew.h>
#include <GL/glut.h>
#include <GL/freeglut.h>

#include "typeDefinition.cuh"
#include "globalVariables.cuh"

// Function protoypes
#include "openGLPrototypes.h"

#include "./common/CudaSafeCall.h"

/*------------------------------------------------------------------------
* Add OpenGL figures/shapes to screen
*------------------------------------------------------------------------
*/

void addFigures(int2 point, float2 pointStim, float2 *trapzAreaCircle,
	float2 *stimAreaCircle, paramVar param,
	int *tip_count, vec5dyn *tip_vector) {

	/*------------------------------------------------------------------------
	* Draw filled circle at the electrode position
	*------------------------------------------------------------------------
	*/

	glPointSize(20.0);
	glEnable(GL_POINT_SMOOTH);
	// glColor3f(1.0,0.0,0.0); // red
	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	// glBlendFunc(GL_CONSTANT_COLOR, GL_ONE_MINUS_DST_COLOR);
	glBegin(GL_POINTS);
	glVertex2f(point.x,point.y);
	glEnd();
	glDisable( GL_BLEND );
	glDisable( GL_POINT_SMOOTH );

	if ( param.reduceSym ) {

		/*------------------------------------------------------------------------
		* Trapezoidal integration domain
		*------------------------------------------------------------------------
		*/

		glLineWidth(2.0);
		// glColor3f(0.0,0.0,1.0); // blue
		glEnable(GL_LINE_SMOOTH);
		glEnable(GL_BLEND);
		// glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
		glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
		glBegin(GL_LINE_STRIP);

		// if ( param.recordTip ) {

		int *tip_pts;
		tip_pts = (int*)malloc(sizeof(int));
		CudaSafeCall(hipMemcpy(tip_pts,tip_count,sizeof(int),hipMemcpyDeviceToHost));

		if (*tip_pts > TIPVECSIZE ) {
		printf("ERROR: NUMBER OF TIP POINTS EXCEEDS tip_vector SIZE\n");
		exit(0);
		}

		vec5dyn *tip_array;
		tip_array = (vec5dyn*)malloc(sizeof(vec5dyn));
		CudaSafeCall(hipMemcpy(tip_array,tip_vector+(*tip_pts-1),
			sizeof(vec5dyn),hipMemcpyDeviceToHost));
		float cx = tip_array[0].x-param.nx/2.f;
		float cy = tip_array[0].y-param.ny/2.f;

		#pragma unroll
		for (int i=0;i<param.nc;i++) {
			glVertex2d(cx+trapzAreaCircle[i].x,cy+trapzAreaCircle[i].y);
		}

		free(tip_pts);
		free(tip_array);


		//   // Plot the circle at the center of the domain
		//   #pragma unroll
		//   for (int i=0;i<param.nc;i++) {
		// 	glVertex2d(trapzAreaCircle[i].x,trapzAreaCircle[i].y);
		//   }
		  
		glEnd();

	}

	if ( param.stimulate ) {

		/*------------------------------------------------------------------------
		* Stimulus/pacing point
		*------------------------------------------------------------------------
		*/

		glPointSize(15.0);
		glEnable(GL_POINT_SMOOTH);
		// glColor3f(1.0,0.0,0.0); // red
		glEnable(GL_BLEND);
		glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
		// glBlendFunc(GL_CONSTANT_COLOR, GL_ONE_MINUS_DST_COLOR);
		glBegin(GL_POINTS);
		glVertex2f(pointStim.x,pointStim.y);
		glEnd();
		glDisable( GL_BLEND );
		glDisable( GL_POINT_SMOOTH );

		/*------------------------------------------------------------------------
		* Stimulus surrounding area
		*------------------------------------------------------------------------
		*/

		glLineWidth(2.0);
		// glColor3f(0.0,0.0,1.0); // blue
		glEnable(GL_LINE_SMOOTH);
		glEnable(GL_BLEND);
		// glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
		glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
		glBegin(GL_LINE_STRIP);
		#pragma unroll
		for (int i=0;i<param.nc;i++) glVertex2d(stimAreaCircle[i].x,stimAreaCircle[i].y);
		glEnd();
	}

}
