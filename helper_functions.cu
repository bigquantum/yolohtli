#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <time.h>

#include "./common/SOIL.h"

#include "typeDefinition.cuh"
// #include "globalVariables.cuh"

#include "hostPrototypes.h"
#include "./common/CudaSafeCall.h"

extern __constant__ int nx_d, ny_d;
extern __device__ vec5dyn tip_vector[TIPVECSIZE];
extern __device__ int tip_count;
extern paramVar param;

/*------------------------------------------------------------------------
* Set optimal block and grid sizes for the CUDA kernel
*------------------------------------------------------------------------
*/

__host__ __device__ int iDivUp(int a, int b) {
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

/*------------------------------------------------------------------------
* Filter algorithms
*------------------------------------------------------------------------
*/

__device__ void push_back3(float3 pt, int *count, float3 *count_vector) {

  count_vector[atomicAdd(count, 1)] = pt;

}

__device__ void push_back5(vec5dyn pt, int *count, vec5dyn *count_vector) {

  count_vector[atomicAdd(count, 1)] = pt;

}

__device__ void plot_field(float ptx, float pty, bool *plot_array) {

  int xIdx = floor(ptx);
  int yIdx = floor(pty);
  plot_array[I2D(nx_d,xIdx,yIdx)] = true;

}

/*------------------------------------------------------------------------
* Compare two quantities
*------------------------------------------------------------------------
*/

__device__ bool equals( REAL a, REAL b, REAL tolerance ) {
    return ( a == b ) ||
      ( ( a <= ( b + tolerance ) ) &&
        ( a >= ( b - tolerance ) ) );
}

/*------------------------------------------------------------------------
* Indices for finite differences
*------------------------------------------------------------------------
*/

__device__ int coord_i(int i) {

  return (int)((i>=0) && (i<nx_d))*i + (int)(i<0)*(-i) + (int)(i>=nx_d)*(2*(nx_d-1)-i);

}

__device__ int coord_j(int j) {

  return (int)((j>=0) && (j<ny_d))*j + (int)(j<0)*(-j) + (int)(j>=ny_d)*(2*(ny_d-1)-j);

}

/*------------------------------------------------------------------------
* Print last tip trajectory points
*------------------------------------------------------------------------
*/

void saveTipLast(int *tip_count, vec5dyn *tip_vector, paramVar *param) {

  int *tip_pts;
  tip_pts = (int*)malloc(sizeof(int));
  CudaSafeCall(hipMemcpy(tip_pts,tip_count,sizeof(int),hipMemcpyDeviceToHost));

  if (*tip_pts > TIPVECSIZE ) {
    printf("ERROR: NUMBER OF TIP POINTS EXCEEDS tip_vector SIZE\n");
    exit(0);
  }

  vec5dyn *tip_array;
  tip_array = (vec5dyn*)malloc((*tip_pts)*sizeof(vec5dyn));
  CudaSafeCall(hipMemcpy(tip_array,tip_vector,(*tip_pts)*sizeof(vec5dyn),hipMemcpyDeviceToHost));

  if ( *tip_pts > 0 ) {
    // Record last tip point
    param->tipx = tip_array[(*tip_pts)-1].x;
    param->tipy = tip_array[(*tip_pts)-1].y;
  } else {
    param->tipx = -1.0f;
    param->tipy = -1.0f;
  }

  free(tip_pts);
  free(tip_array);

}

/*------------------------------------------------------------------------
* Linear interpolation
*------------------------------------------------------------------------
*/

float host_lerp(float v0, float v1, float t) {
  return (1 - t) * v0 + t * v1;
}

__device__ inline REAL my_lerp(REAL v0, REAL v1, REAL t) {
    //return (1.f-t)*v0 + t*v1;
    return fma(t, v1, fma(-t, v0, v0));
}

/*------------------------------------------------------------------------
* Swap array pointers
*------------------------------------------------------------------------
*/

void swap(float* &a, float* &b) {
  float *temp = a;
  a = b;
  b = temp;
}

void swapSoA(stateVar *A, stateVar *B) {
    stateVar temp = *A;
    *A = *B;
    *B = temp;
}

/*------------------------------------------------------------------------
* Sign function
*------------------------------------------------------------------------
*/

__device__ int sign(REAL x) { 

  int t = x < 0.0 ? -1 : 0;

  return x > 0.0 ? 1 : t;

}

/*------------------------------------------------------------------------
* Take screenshot
*------------------------------------------------------------------------
*/

void screenShot(int w, int h) {
	
	time_t t = time(NULL);
	struct tm tm = *localtime(&t);

    char name[100];
    sprintf(name, "./DATA/screenshots/figure_%d-%d-%d_%d-%d-%d.bmp", 
    	tm.tm_year + 1900, tm.tm_mon + 1, 
    	tm.tm_mday, tm.tm_hour, tm.tm_min, tm.tm_sec);
      /* save a screenshot */
      // sudo apt-get install libsoil-dev
    SOIL_save_screenshot(name,
                         SOIL_SAVE_TYPE_BMP,
                         0, 0, w, h);

}

/*------------------------------------------------------------------------
* Press Enter key to confirm
*------------------------------------------------------------------------
*/

void pressEnterKey(void) {
  // Ask for ENTER key
  printf("Press [Enter] key to continue\n");
  printf("[Ctrl]+[C] to terminate program.\n");
  while(getchar()!='\n'); // option TWO to clean stdin
  getchar(); // wait for ENTER
}

/*------------------------------------------------------------------------
* Conduction block
*------------------------------------------------------------------------
*/

void conductionBlock(int memSize, bool counterclock, bool clock1,
  stateVar g_h, stateVar g_present_d) {

  int i, j, idx;

  CudaSafeCall(hipMemcpy(g_h.u, g_present_d.u, memSize,
    hipMemcpyDeviceToHost));

  if (counterclock) {
    for (j=0;j<(param.ny/2+70);j++) {
    // for (j=0;j<(param.ny/2-10);j++) {
    // for (j=0;j<(param.ny/2+50);j++) {
      for (i=0;i<(param.nx);i++) {
        idx = i + param.nx * j;
        g_h.u[idx] = 0.0;
        }
      }

    }

  if (clock1) {

    for (j=0;j<param.ny;j++) {
      for (i=(param.nx/2);i<(param.nx);i++) {
        idx = i + param.nx * j;
        g_h.u[idx] = 0.0;
        }
      }

    }

  CudaSafeCall(hipMemcpy(g_present_d.u, g_h.u, memSize,
    hipMemcpyHostToDevice));

}

/*------------------------------------------------------------------------
* Checks for the voltage level
*------------------------------------------------------------------------
*/

bool isThereFib(REAL *voltage, paramVar param) {

  REAL v = 0.0;
  for (int i=floor(9*param.wnx/10.0);i<param.wnx;i++) {
    v += voltage[i];
  }

  return v>0.5 ? true : false;

}